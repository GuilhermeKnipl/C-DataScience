
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *result, int n) {
    int i = threadIdx.x;
    if (i < n)
        result[i] = a[i] + b[i];
}

int main() {
    int n = 4;
    int size = n * sizeof(int);

    // Host memory
    int h_a[] = {1, 2, 3, 4};
    int h_b[] = {10, 20, 30, 40};
    int h_result[4];

    // Device memory
    int *d_a, *d_b, *d_result;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_result, size);

    // Copy to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch 1 block with 4 threads
    vector_add<<<1, 4>>>(d_a, d_b, d_result, n);

    // Copy result back to host
    hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < n; i++)
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_result[i]);

    // Free device memory
    hipFree(d_a); hipFree(d_b); hipFree(d_result);

    return 0;
}
